#include "hip/hip_runtime.h"
#include <optix.h>
#include <sutil/vec_math.h>
#include "optixScan.h"

__global__ void kGenAABB_t (
      double3* points,
      double radius,
      unsigned int N,
      OptixAabb* aabb
) {
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= N) return;

  float3 center = {points[particleIndex].x, points[particleIndex].y, points[particleIndex].z};
  float3 m_min = center - radius;
  float3 m_max = center + radius;
  aabb[particleIndex] =
  {
    m_min.x, m_min.y, m_min.z,
    m_max.x, m_max.y, m_max.z
  };
}

extern "C" void kGenAABB(double3* points, double width, unsigned int numPrims, OptixAabb* d_aabb, int epi) {
  unsigned int threadsPerBlock = 64;
  unsigned int numOfBlocks = numPrims / threadsPerBlock + 1;

  kGenAABB_t <<<numOfBlocks, threadsPerBlock>>> (
      points,
      width + epi,
      numPrims,
      d_aabb
     );
}
